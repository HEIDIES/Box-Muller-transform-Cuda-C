#include "hip/hip_runtime.h"
//
// Created by heidies on 7/2/18.
//

#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <opencv2/opencv.hpp>
#include <cstring>

using namespace std;
using namespace cv;

#define pi 3.1415926
#define CHECK(call){                                                                                                      \
    const hipError_t error = call;                                                                                         \
    if (error != hipSuccess){                                                                                             \
        cout << "Error: " << __FILE__ << ":" << __LINE__ << endl;                                                           \
        cout << "code: " << error << "reason: " << hipGetErrorString(error) << endl;                                          \
        exit(-10 * error);                                                                                                \
    }                                                                                                                    \
}

// Generate a uniform random list
void initialRandomNum(float *ip, int size){
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size ; ++ i){
        ip[i] = rand() / RAND_MAX;
    }
}

// Add white noise N(0, sigma) to origin image
// Box-Muller transform
__global__ void addWhiteNoise(float *Img, float *randList, const int imgWidth, const int imgHeight, const float sigma){
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;

    int z0_idx = iy * 2 * imgWidth + ix;
    int z1_idx = iy * (2 * imgWidth + 1) + ix;
    if(z0_idx < imgHeight) {
        Img[z0_idx] += sigma * cos(2 * pi * randList[z0_idx]) * sqrt(-2 * log * randList[z1_idx]);
        Img[z0_idx] = Img[z0_idx] < 0? 0.0 : Img[z0_idx] > 255.0? 255.0, Img[z0_idx];
    }
    if(z1_idx < imgHeight) {
        Img[z1_idx] += sigma * sin(2 * pi * randList[z0_idx]) * sqrt(-2 * log * randList[z1_idx]);
        Img[z1_idx] = Img[z1_idx] < 0? 0.0 : Img[z0_idx] > 255.0? 255.0, Img[z1_idx];
    }
}

// test
int main(int argc, char **argv){
    string filePath = "00001.jpg";
    Mat Img = (float)imread(filePath, CV_LOAD_IMAGE_GRAYSCALE);
    dim3 block(32, 32);
    dim3 grid((Img.cols + block.x - 1) / block.x, (Img.rows / 2 + block.y - 1) / block.y);
    float
    return 0;
}


